#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>

#define N 1024  // Matrix size (NxN)

// CUDA Kernel for matrix addition
__global__ void matrixAdd(int *A, int *B, int *C, int N) {
    // Calculate the index of the thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we stay within bounds
    if (row < N && col < N) {
        int index = row * N + col;
        C[index] = A[index] + B[index];
    }
}

int main() {
    int *h_A, *h_B, *h_C;  // Host matrices
    int *d_A, *d_B, *d_C;  // Device matrices

    // Allocate memory for matrices on the host
    h_A = (int*)malloc(N * N * sizeof(int));
    h_B = (int*)malloc(N * N * sizeof(int));
    h_C = (int*)malloc(N * N * sizeof(int));

    // Initialize matrices with random values
    srand(time(0));
    for (int i = 0; i < N * N; i++) {
        h_A[i] = rand() % 100;
        h_B[i] = rand() % 100;
    }

    // Allocate memory for matrices on the device (GPU)
    hipMalloc((void**)&d_A, N * N * sizeof(int));
    hipMalloc((void**)&d_B, N * N * sizeof(int));
    hipMalloc((void**)&d_C, N * N * sizeof(int));

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Define block and grid size
    dim3 threadsPerBlock(16, 16);  // 16x16 block
    dim3 numBlocks((N + 15) / 16, (N + 15) / 16);  // Grid size

    // Launch the kernel
    matrixAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy the result matrix back to the host
    hipMemcpy(h_C, d_C, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Optionally, print the result (for small N)
//     /*
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << h_C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }
//     */

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
